#include "hip/hip_runtime.h"
#include "ex1.h"

#define HIST_SIZE 256
#define HIST_SIZE_MEMORY 256 * sizeof(int)
#define IMG_SIZE (IMG_HEIGHT*IMG_WIDTH)

// Make the given histogram a CDF array
__device__ void prefix_sum(int arr[], int arr_size) { //TODO: switch to parallel
    int sum = 0;
    for(int i=0;i<arr_size;i++) {
        sum += arr[i];
        arr[i] = sum;
    }
    return;
}

// __device__ void map_calculation(int arr[], uchar* map, int arr_size) {
    
//     return;
// }

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__ 
void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

// Zeros the maps array of a given context
__device__ void reset_maps_array(uchar *maps, int *hist) {
    for (int i=0;i<TILE_COUNT * TILE_COUNT * HIST_SIZE;i++) {

                maps[i] = (unsigned char)0;
                hist[i] = (int)0;

    }


}




__global__ void process_image_kernel(uchar *all_in, uchar *all_out, uchar *maps,int *hist) {
    int tid=threadIdx.x;


    if(tid == 0) {
        reset_maps_array(maps, hist);
    }
    __syncthreads();

    // Each thread would calculate the histogram contribution
    // of a single row of length T in a specific tile.

    // Calculate offset of this thread in the image pixels array
    int tileID = tid / 8;
    int rowInTile = tid % 8;

    int numberOfTilesInRowImg = IMG_WIDTH / TILE_WIDTH;
    int gridRow = tileID / numberOfTilesInRowImg;
    int gridCol = tileID % numberOfTilesInRowImg;

    int left = TILE_WIDTH*gridCol;
    int right = TILE_WIDTH*(gridCol+1) - 1;
    int top = TILE_WIDTH*gridRow;
    int bottom = TILE_WIDTH*(gridRow+1) - 1;

    for (int y = 0; y < 8; y++)
    {
        for (int x=left; x<=right; x++) {
            uchar *row = all_in + (top + rowInTile*8 + y) * IMG_WIDTH;
            int val = row[x];
            atomicAdd((&(hist[(gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + val])),1);
        }
    }

    __syncthreads();

    // Make the histogram into a map by only the first thread in every tile
    if(rowInTile == 0) { //TODO: make it parallel
        // Now make the histogram a CDF, by running prefix_sum
        prefix_sum(&hist[(gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE],HIST_SIZE);
        
        // Perform map calculation for each tile        
        // Make the given CDF array a map, using the given definition
        for(int i=0;i<HIST_SIZE;i++) {
            maps[(gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + i] = (float(hist[(gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + i]) * 255) / (TILE_WIDTH*TILE_WIDTH);
            //hist[(gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + i] = (float(hist[(gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + i]) * 255) / (TILE_WIDTH*TILE_WIDTH);
        }
    }

    // __syncthreads();
    // // Now we copy hist values to maps
    // for (int i=0;i<TILE_COUNT;i++) {
    //     for (int j=0;j<TILE_COUNT;j++) {
    //         for (int n=0;n<HIST_SIZE;n++) {
    //                 maps[i * TILE_COUNT * HIST_SIZE + j * HIST_SIZE + n] = (unsigned char)(hist[i * TILE_COUNT * HIST_SIZE + j * HIST_SIZE + n]);
    //         }
    //     }
    // }

    __syncthreads();

    
    interpolate_device(maps, all_in, all_out);
    return; 
}


/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context {
    uchar *all_in;
    uchar *all_out;
    uchar *maps;
    int *hist;
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context *task_serial_init()
{
    auto context = new task_serial_context;

    CUDA_CHECK(hipMalloc((void**) &(context->all_in), IMG_WIDTH * IMG_WIDTH));
    CUDA_CHECK(hipMalloc((void**) &(context->all_out), IMG_WIDTH * IMG_WIDTH));
    CUDA_CHECK(hipMalloc((void**) &(context->maps), TILE_COUNT * TILE_COUNT * HIST_SIZE));
    CUDA_CHECK(hipMalloc((void**) &(context->hist), TILE_COUNT * TILE_COUNT * HIST_SIZE * sizeof(int)));



    return context;
}



/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_in, uchar *images_out) //TODO: why one certain context for all images?
{
    //in a for loop:
    //   1. copy the relevant image from images_in to the GPU memory you allocated
    //   2. invoke GPU kernel on this image
    //   3. copy output from GPU memory to relevant location in images_out_gpu_serial


    
    // calculate the number of threads in one image
    int threads_in_block = (TILE_WIDTH * TILE_COUNT * TILE_COUNT) / 8;

    for (int i=0;i<N_IMAGES;i++) {
        CUDA_CHECK((hipMemcpy(context->all_in, (images_in + i * IMG_SIZE), IMG_SIZE, hipMemcpyHostToDevice)));

        process_image_kernel<<<1,threads_in_block>>>(context->all_in,context->all_out,context->maps,context->hist);

        CUDA_CHECK(hipGetLastError());
        
        CUDA_CHECK((hipMemcpy((images_out + i * IMG_SIZE), context->all_out, IMG_SIZE, hipMemcpyDeviceToHost)));
    }

}

/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    CUDA_CHECK(hipFree((void**)(context->all_in)));
    CUDA_CHECK(hipFree((void**)(context->all_out)));
    CUDA_CHECK(hipFree((void**)(context->maps)));
    CUDA_CHECK(hipFree((void**)(context->hist)));

    free(context);
}


/////////////////BULK PROCESS/////////////////
__global__ void bulk_process_image_kernel(uchar *all_in, uchar *all_out, uchar *maps,int *hist) { //TODO:
    int tid=threadIdx.x;
    int bid=blockIdx.x;


    //int hist[TILE_COUNT][TILE_COUNT][HIST_SIZE];

    // Each thread would calculate the histogram contribution
    // of a single row of length T in a specific tile.

    // Calculate offset of this thread in the image pixels array
    int imgID = bid;
    int tileID = tid / 8;
    int rowInTile = tid % 8;

    int numberOfTilesInRowImg = IMG_WIDTH / TILE_WIDTH;
    int gridRow = tileID / numberOfTilesInRowImg;
    int gridCol = tileID % numberOfTilesInRowImg;


    int left = TILE_WIDTH*gridCol;
    int right = TILE_WIDTH*(gridCol+1) - 1;
    int top = TILE_WIDTH*gridRow;
    int bottom = TILE_WIDTH*(gridRow+1) - 1;

      if(tid == 0) {
        reset_maps_array(&maps[imgID*numberOfTilesInRowImg*numberOfTilesInRowImg*HIST_SIZE], &hist[imgID*numberOfTilesInRowImg*numberOfTilesInRowImg*HIST_SIZE]);
    }
    __syncthreads();

    for (int y = 0; y < 8; y++)
    {
        for (int x=left; x<=right; x++) {
            uchar *row = all_in + imgID*IMG_SIZE + (top + rowInTile*8 + y) * IMG_WIDTH ;
            int val = row[x];
            atomicAdd((&(hist[imgID*numberOfTilesInRowImg*numberOfTilesInRowImg*HIST_SIZE + (gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + val])),1);
        }
    }

    __syncthreads();

    // Make the histogram into a map by only the first thread in every tile
    if(rowInTile == 0) {
        // Now make the histogram a CDF, by running prefix_sum
        prefix_sum(&hist[imgID*numberOfTilesInRowImg*numberOfTilesInRowImg*HIST_SIZE + (gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE],HIST_SIZE);
        
        // Perform map calculation for each tile        
        // Make the given CDF array a map, using the given definition
        for(int i=0;i<HIST_SIZE;i++) {
            maps[imgID*numberOfTilesInRowImg*numberOfTilesInRowImg*HIST_SIZE + (gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + i] = 
            (float(hist[imgID*numberOfTilesInRowImg*numberOfTilesInRowImg*HIST_SIZE + (gridRow * numberOfTilesInRowImg + gridCol) * HIST_SIZE + i]) * 255) / (TILE_WIDTH*TILE_WIDTH);
            
        }
    }
 
    __syncthreads();

    interpolate_device(&maps[imgID*numberOfTilesInRowImg*numberOfTilesInRowImg*HIST_SIZE], (all_in + imgID*IMG_SIZE), (all_out + imgID*IMG_SIZE));

    __syncthreads();

    return; 
}


/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context {
    uchar *all_in;
    uchar *all_out;
    uchar *maps;
    int *hist;};

/* Allocate GPU memory for all the input images, output images, and maps.
 * 
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;

    CUDA_CHECK(hipMalloc((void**) &(context->all_in), IMG_WIDTH * IMG_WIDTH * N_IMAGES));
    CUDA_CHECK(hipMalloc((void**) &(context->all_out), IMG_WIDTH * IMG_WIDTH * N_IMAGES));
    CUDA_CHECK(hipMalloc((void**) &(context->maps), TILE_COUNT * TILE_COUNT * HIST_SIZE * N_IMAGES));
    // CUDA_CHECK(hipMemset((context->maps),(unsigned char)0,TILE_COUNT * TILE_COUNT * HIST_SIZE * N_IMAGES));
    CUDA_CHECK(hipMalloc((void**) &(context->hist), TILE_COUNT * TILE_COUNT * HIST_SIZE * sizeof(int) * N_IMAGES));
    // CUDA_CHECK(hipMemset((context->hist),0,TILE_COUNT * TILE_COUNT * HIST_SIZE * sizeof(int) * N_IMAGES));


    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_in, uchar *images_out)
{
    // calculate the number of threads in one image
    int threads_in_block = (TILE_WIDTH * TILE_COUNT * TILE_COUNT) / 8;

    CUDA_CHECK((hipMemcpy(context->all_in, images_in, (N_IMAGES * IMG_SIZE), hipMemcpyHostToDevice)));

    bulk_process_image_kernel<<<N_IMAGES,threads_in_block>>>(context->all_in,context->all_out,context->maps,context->hist);

    CUDA_CHECK((hipMemcpy(images_out, context->all_out, (N_IMAGES * IMG_SIZE), hipMemcpyDeviceToHost)));
    
}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    CUDA_CHECK(hipFree((void**)(context->all_in)));
    CUDA_CHECK(hipFree((void**)(context->all_out)));
    CUDA_CHECK(hipFree((void**)(context->maps)));
    CUDA_CHECK(hipFree((void**)(context->hist)));

    free(context);
}
